
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define A 1
#define G 2
#define C 3
#define T 4

using namespace std;

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

__global__ void f_scoreSequence(float* query, float* subject, float* scoringMatrix, int width, int height) {
    // Do the scoring
    int substitutionMatrix[2] = {SEQ_EQUAL, SEQ_DIFF};
    
    register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    //register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    float maxScore = 0;
    for (int i = 1; i < (height + 1); i++) {
        for (int j = 1; j < (width + 1); j++) {
            float score = 0;
            
            score = max(score, scoringMatrix[(i * (width + 1)) + j - 1] - GAP_PENALTY);
            score = max(score, scoringMatrix[((i - 1) * (width + 1)) + j] - GAP_PENALTY);
            
            int similarityScore = 0;
            if (query[i - 1] == subject[32*j - 1 + xIndex]) similarityScore = substitutionMatrix[0];
            else similarityScore = substitutionMatrix[1];
            
            score = max(score, scoringMatrix[((i - 1) * (width + 1)) + j - 1] + similarityScore);
            
            maxScore = max(maxScore, score);
                    
            scoringMatrix[(i * (width + 1)) + j] = score;
        }
    }
}

int main( int argc, char *argv[] ) {
    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }

    string querySequence = argv[1];
    
    // Parse query file
    ifstream datafile;
    datafile.open(argv[2]);
    
    string temp;
    vector<string> subjectSequences;
    while (datafile >> temp) {
       subjectSequences.push_back(temp);
    }
    
    // Just do the first 32 elements for a test
    int largestSubjectLength = subjectSequences[31].length();
    
    datafile.close();
    
    // alloc memory on GPU
    float* d_input_query;
    hipMallocManaged((void**) &d_input_query, querySequence.length() * sizeof(float));
    
    float* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (largestSubjectLength * 32) * sizeof(float));
    
    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((querySequence.length() + 1) * (largestSubjectLength + 1) * 32) * sizeof(float));
    
    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < querySequence.length();i++) { // Pad to nearest 8 eventually here
        switch(querySequence[i])
        {
            case 'A': { d_input_query[i] = A;
                        break;
                    }
            case 'G': { d_input_query[i] = G;
                        break;
                    }
            case 'C': { d_input_query[i] = C;
                        break;
                    }
            case 'T': { d_input_query[i] = T;
                        break;
                    }
        }
    }
    
    for (int j = 0; j < 32; j++) {
        for (int i = 0; i < largestSubjectLength; i++) { // Will need to pad here
            switch(subjectSequences[j][i])
            {
                case 'A': { d_input_subject[i*32 + j] = A;
                            break;
                        }
                case 'G': { d_input_subject[i*32 + j] = G;
                            break;
                        }
                case 'C': { d_input_subject[i*32 + j] = C;
                            break;
                        }
                case 'T': { d_input_subject[i*32 + j] = T;
                            break;
                        }
            }
        }
    }
    
    // Call GPU
    dim3 block(32, 1);
    dim3 grid(1, 1);
 
    f_scoreSequence<<<grid, block>>>(d_input_query, d_input_subject, d_output_scoring, largestSubjectLength, querySequence.length());
    
    hipDeviceSynchronize();
    
    // Print results
    int subject = 0;
    string seqA = querySequence;
    string seqB = subjectSequences[subject];
    
    cout << "    ";
    for (int j = 0; j < (seqB.length() + 1); j++) {
        cout << seqB[j] << " "; 
    }
    cout << endl;
    
    for (int i = 0; i < (seqA.length() + 1); i++) {
        if (i != 0) cout << seqA[i - 1] << " ";
        else cout << "  ";
        for (int j = 0; j < (seqB.length() + 1); j++) {
            cout << d_output_scoring[i * (seqB.length() + 1) + j] << " "; 
        }
        cout << endl;
    }
    
    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_output_scoring);
    hipDeviceReset();
}
