#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <math.h>
#include <sstream>
#include <algorithm>
#include "boost/program_options.hpp"

#include "FASTAParsers.h"
#include "SWSolver.h"

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define LENGTH_THRESHOLD 100

#define A 0
#define R 1
#define N 2
#define D 3
#define C 4
#define Q 5
#define E 6
#define G 7
#define H 8
#define I 9
#define L 10
#define K 11
#define M 12
#define F 13
#define P 14
#define S 15
#define T 16
#define W 17
#define Y 18
#define V 19
#define B 20
#define J 21
#define Z 22
#define X 23
#define STAR 24

#define BLOCK_Y_DIM 32.0

int blosum50[25][25] = {
//        A  R  N  D  C  Q  E  G  H  I  L  K  M  F  P  S  T  W  Y  V  B  J  Z  X  *
/* A */ { 5,-2,-1,-2,-1,-1,-1, 0,-2,-1,-2,-1,-1,-3,-1, 1, 0,-3,-2, 0,-2,-2,-1,-1,-5},
/* R */ {-2, 7,-1,-2,-4, 1, 0,-3, 0,-4,-3, 3,-2,-3,-3,-1,-1,-3,-1,-3,-1,-3, 0,-1,-5},
/* N */ {-1,-1, 7, 2,-2, 0, 0, 0, 1,-3,-4, 0,-2,-4,-2, 1, 0,-4,-2,-3, 5,-4, 0,-1,-5},
/* D */ {-2,-2, 2, 8,-4, 0, 2,-1,-1,-4,-4,-1,-4,-5,-1, 0,-1,-5,-3,-4, 6,-4, 1,-1,-5 },
/* C */ {-1,-4,-2,-4,13,-3,-3,-3,-3,-2,-2,-3,-2,-2,-4,-1,-1,-5,-3,-1,-3,-2,-3,-1,-5 }, 
/* Q */ {-1, 1, 0, 0,-3, 7, 2,-2, 1,-3,-2, 2, 0,-4,-1, 0,-1,-1,-1,-3, 0,-3, 4,-1,-5 },  
/* E */ {-1, 0, 0, 2,-3, 2, 6,-3, 0,-4,-3, 1,-2,-3,-1,-1,-1,-3,-2,-3, 1,-3, 5,-1,-5 }, 
/* G */ { 0,-3, 0,-1,-3,-2,-3, 8,-2,-4,-4,-2,-3,-4,-2, 0,-2,-3,-3,-4,-1,-4,-2,-1,-5 },
/* H */ {-2, 0, 1,-1,-3, 1, 0,-2,10,-4,-3, 0,-1,-1,-2,-1,-2,-3, 2,-4, 0,-3, 0,-1,-5 },
/* I */ {-1,-4,-3,-4,-2,-3,-4,-4,-4, 5, 2,-3, 2, 0,-3,-3,-1,-3,-1, 4,-4, 4,-3,-1,-5 },
/* L */ {-2,-3,-4,-4,-2,-2,-3,-4,-3, 2, 5,-3, 3, 1,-4,-3,-1,-2,-1, 1,-4, 4,-3,-1,-5 },
/* K */ {-1, 3, 0,-1,-3, 2, 1,-2, 0,-3,-3, 6,-2,-4,-1, 0,-1,-3,-2,-3, 0,-3, 1,-1,-5 },
/* M */ {-1,-2,-2,-4,-2, 0,-2,-3,-1, 2, 3,-2, 7, 0,-3,-2,-1,-1, 0, 1,-3, 2,-1,-1,-5 },
/* F */ {-3,-3,-4,-5,-2,-4,-3,-4,-1, 0, 1,-4, 0, 8,-4,-3,-2, 1, 4,-1,-4, 1,-4,-1,-5 },
/* P */ {-1,-3,-2,-1,-4,-1,-1,-2,-2,-3,-4,-1,-3,-4,10,-1,-1,-4,-3,-3,-2,-3,-1,-1,-5 },
/* S */ { 1,-1, 1, 0,-1, 0,-1, 0,-1,-3,-3, 0,-2,-3,-1, 5, 2,-4,-2,-2, 0,-3, 0,-1,-5 },
/* T */ { 0,-1, 0,-1,-1,-1,-1,-2,-2,-1,-1,-1,-1,-2,-1, 2, 5,-3,-2, 0, 0,-1,-1,-1,-5 },
/* W */ {-3,-3,-4,-5,-5,-1,-3,-3,-3,-3,-2,-3,-1, 1,-4,-4,-3,15, 2,-3,-5,-2,-2,-1,-5 },
/* Y */ {-2,-1,-2,-3,-3,-1,-2,-3, 2,-1,-1,-2, 0, 4,-3,-2,-2, 2, 8,-1,-3,-1,-2,-1,-5 },
/* V */ { 0,-3,-3,-4,-1,-3,-3,-4,-4, 4, 1,-3, 1,-1,-3,-2, 0,-3,-1, 5,-3, 2,-3,-1,-5 },
/* B */ {-2,-1, 5, 6,-3, 0, 1,-1, 0,-4,-4, 0,-3,-4,-2, 0, 0,-5,-3,-3, 6,-4, 1,-1,-5 },
/* J */ {-2,-3,-4,-4,-2,-3,-3,-4,-3, 4, 4,-3, 2, 1,-3,-3,-1,-2,-1, 2,-4, 4,-3,-1,-5 },
/* Z */ {-1, 0, 0, 1,-3, 4, 5,-2, 0,-3,-3, 1,-1,-4,-1, 0,-1,-2,-2,-3, 1,-3, 5,-1,-5 }, 
/* X */ {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-5 },
/* * */ {-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5, 1 }
};

// rechecking values - complete: ABCDEFGHIJKLMN PQRST VWXYZ
int blosum50_alpha[25][25] = {
//        A  B  C  D  E  F  G  H  I  J  K  L  M  N  P  Q  R  S  T  V  W  X  Y  Z  *
/* A */ { 5,-2,-1,-2,-1,-3, 0,-2,-1,-2,-1,-2,-1,-1,-1,-1,-2, 1, 0, 0,-3,-1,-2,-1,-5 },
/* B */ {-2, 6,-3, 6, 1,-4,-1, 0,-4,-4, 0,-4,-3, 5,-2, 0,-1, 0, 0,-3,-5,-1,-3, 1,-5 },
/* C */ {-1,-3,13,-4,-3,-2,-3,-3,-2,-2,-3,-2,-2,-2,-4,-3,-4,-1,-1,-1,-5,-1,-3,-3,-5 },
/* D */ {-2, 6,-4, 8, 2,-5,-1,-1,-4,-4,-1,-4,-4, 2,-1, 0,-2, 0,-1,-4,-5,-1,-3, 1,-5 },
/* E */ {-1, 1,-3, 2, 6,-3,-3, 0,-4,-3, 1,-3,-2, 0,-1, 2, 0,-1,-1,-3,-3,-1,-2, 5,-5 }, 
/* F */ {-3,-4,-2,-5,-3, 8,-4,-1, 0, 1,-4, 1, 0,-4,-4,-4,-3,-3,-2,-1, 1,-1, 4,-4,-5 },  
/* G */ { 0,-1,-3,-1,-3,-4, 8,-2,-4,-4,-2,-4,-3, 0,-2,-2,-3, 0,-2,-4,-3,-1,-3,-2,-5 }, 
/* H */ {-2, 0,-3,-1, 0,-1,-2,10,-4,-3, 0,-3,-1, 1,-2, 1, 0,-1,-2,-4,-3,-1, 2, 0,-5 },
/* I */ {-1,-4,-2,-4,-4, 0,-4,-4, 5, 4,-3, 2, 2,-3,-3,-3,-4,-3,-1, 4,-3,-1,-1,-3,-5 },
/* J */ {-2,-4,-2,-4,-3, 1,-4,-3, 4, 4,-3, 4, 2,-4,-3,-3,-3,-3,-1, 2,-2,-1,-1,-3,-5 },
/* K */ {-1, 0,-3,-1, 1,-4,-2, 0,-3,-3, 6,-3,-2, 0,-1, 2, 3, 0,-1,-3,-3,-1,-2, 1,-5 },
/* L */ {-2,-4,-2,-4,-3, 1,-4,-3, 2, 4,-3, 5, 3,-4,-4,-2,-3,-3,-1, 1, 2,-1,-1,-3,-5 },
/* M */ {-1,-3,-2,-4,-2, 0,-3,-1, 2, 2,-2, 3, 7,-2,-3, 0,-2,-2,-1, 1,-1,-1, 0,-1,-5 },
/* N */ {-1, 5,-2, 2, 0,-4, 0, 1,-3,-4, 0,-4,-2, 7,-2, 0,-1, 1, 0,-3,-4,-1,-2, 0,-5 },
/* P */ {-1,-2,-4,-1,-1,-4,-2,-2,-3,-3,-1,-4,-3,-2,10,-1,-3,-1,-1,-3,-4,-1,-3,-1,-5 },
/* Q */ {-1, 0,-3, 0, 2,-4,-2, 1,-3,-3, 2,-2, 0, 0,-1, 7, 1, 0,-1,-3,-1,-1,-1, 4,-5 },
/* R */ {-2,-1,-4,-2, 0,-3,-3, 0,-4,-3, 3,-3,-2,-1,-3, 1, 7,-1,-1,-3,-3,-1,-1, 0,-5 },
/* S */ { 1, 0,-1, 0,-1,-3, 0,-1,-3,-3, 0,-3,-2, 1,-1, 0,-1, 5, 2,-2,-4,-1,-2, 0,-5 },
/* T */ { 0, 0,-1,-1,-1,-2,-2,-2,-1,-1,-1,-1,-1, 0,-1,-1,-1, 2, 5, 0,-3,-1,-2,-1,-5 },
/* V */ { 0,-3,-1,-4,-3,-1,-4,-4, 4, 2,-3, 1, 1,-3,-3,-3,-3,-2, 0, 5,-3,-1,-1,-3,-5 },
/* W */ {-3,-5,-5,-5,-3, 1,-3,-3,-3,-2,-3,-2,-1,-4,-4,-1,-3,-4,-3,-3,15,-1, 2,-2,-5 },
/* X */ {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-5 },
/* Y */ {-2,-3,-3,-3,-2, 4,-3, 2,-1,-1,-2,-1, 0,-2,-3,-1,-1,-2,-2,-1, 2,-1, 8,-2,-5 }, 
/* Z */ {-1, 1,-3, 1, 5,-4,-2, 0,-3,-3, 1,-3,-1, 0,-1, 4, 0, 0,-1,-3,-2,-1,-2, 5,-5 },
/* * */ {-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5, 1 }
};

using namespace std;
namespace po = boost::program_options;

__constant__ char constQuery[1024];
__constant__ int constSubstitutionMatrix[625];

char convertStringToChar(char character) {
    switch(character)
    {
        case 'A': { return 'A'; }
        case 'R': { return 'R'; }
        case 'N': { return 'N'; }
        case 'D': { return 'D'; }
        case 'C': { return 'C'; }
        case 'Q': { return 'Q'; }
        case 'E': { return 'E'; }
        case 'G': { return 'G'; }
        case 'H': { return 'H'; }
        case 'I': { return 'I'; }
        case 'L': { return 'L'; }
        case 'K': { return 'K'; }
        case 'M': { return 'M'; }
        case 'F': { return 'F'; }
        case 'P': { return 'P'; }
        case 'S': { return 'S'; }
        case 'T': { return 'T'; }
        case 'W': { return 'W'; }
        case 'Y': { return 'Y'; }
        case 'V': { return 'V'; }
        case 'B': { return 'B'; }
        case 'J': { return 'J'; }
        case 'Z': { return 'Z'; }
        case 'X': { return 'X'; }
    }
    return '*';
}

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequence(char* subject, float* scoringMatrix, float* maxScoreList, int width, int height, int numSubjects) {

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (yIndex == 0) {
        //printf("GPU: %d %f\n", constSubstitutionMatrix[24], constQuery[0]);
    }
    
    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);
				
				int similarityScore = 0;
				int query = (int)constQuery[i-1];
				int subjectValue = (int)subject[width*yIndex + j -1];
				int searchMatrix;
				
				// first determine how to search the constSubstitutionMatrix
				// have both in V-*, 1 in V-Z 1 in P-T (vice versa flip them), 1 in V-Z 1 in A-N (vice versa flip them),
				// both in P-T, 1 in P-T 1 in A-N (vice versa flip them), both in A-N
				// 6 possibilities after collapsing vice versas
				if ((query > 85 || query == 42) && (subjectValue > 85 || subjectValue == 42)) // both in V-*
					searchMatrix = 1;
				else if ((query > 85 || query == 42) && (subjectValue > 79)) // query in V-*, subjectValue in P-T
					searchMatrix = 2;
				else if ((query > 79) && (subjectValue > 85 || subjectValue == 42)) {// query in P-T, subjectValue in V-*
					int temp = query;
					query = subjectValue;
					subjectValue = temp;
					searchMatrix = 2;
				}
				else if (query > 85 || query == 42) // query in V-*, subjectValue in A-N
					searchMatrix = 3;
				else if (subjectValue > 85 || subjectValue == 42) { // query in A-N, subjectValue in V-*
					int temp = query;
					query = subjectValue;
					subjectValue = temp;
					searchMatrix = 3;
				}

				else if (query > 79 && subjectValue > 79) // both in P-T
					searchMatrix = 4;
				else if (query > 79) // query in P-T, subjectValue in A-N
					searchMatrix = 5;
				else if (subjectValue > 79) { // query in A-N, subjectValue in P-T
					int temp = query;
					query = subjectValue;
					subjectValue = temp;
					searchMatrix = 5;
				}

				else // both in A-N
					searchMatrix = 6;

				// based on the searchMatrix value, use a switch case and calculate the similarityScore
				// if value is * (42), result is either -5 (not similar) or 1 (similar)
				switch(searchMatrix)
				{
					case 1: {
						if (query == 42 && subjectValue == 42) similarityScore = 1;
						else if ((query == 42 && subjectValue != 42) || (query != 42 && subjectValue == 42)) similarityScore = -5;
						else similarityScore = constSubstitutionMatrix[((query-67) * 25) + subjectValue-67]; 
						break; }

					case 2: {
						if (query == 42) similarityScore = -5;
						else similarityScore = constSubstitutionMatrix[((query-67) * 25) + subjectValue-66];
						break; }
				
					case 3: {
						if (query == 42) similarityScore = -5;
						else similarityScore = constSubstitutionMatrix[((query-67) * 25) + subjectValue-65];
						break; }
			
					case 4: {
						similarityScore = constSubstitutionMatrix[((query-66) * 25) + subjectValue-66];
						break; }

					case 5: {
						similarityScore = constSubstitutionMatrix[((query-66) * 25) + subjectValue-65];
						break; }
			
					case 6: {
						similarityScore = constSubstitutionMatrix[((query-65) * 25) + subjectValue-65];
						break; }
				}
					
                //int similarityScore = constSubstitutionMatrix[((int)constQuery[i - 1] * 25) + (int)subject[width*yIndex + j - 1]];
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);

                maxScore = max(maxScore, score);

                scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequenceCoalesced(float* subject, float* scoringMatrix, float* maxScoreList, 
                int width /*largestSubjectLength*/, int height /*querySequence.length()*/, int numSubjects) {
    
    int substitutionMatrix[2] = {3, -3};

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use map for different offsets (Change the width)
    int blockOffset = (blockIdx.y * blockDim.y)*(width + 1)*(height + 1);

    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * i)] - GAP_PENALTY);
                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] - GAP_PENALTY);

                int similarityScore = 0;

                if (constQuery[i - 1] == subject[threadIdx.y + ((j - 1) * blockDim.y)]) similarityScore = substitutionMatrix[0];
                else similarityScore = substitutionMatrix[1];

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] + similarityScore);

                maxScore = max(maxScore, score);
                
                scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * i)] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

int main( int argc, char *argv[] ) {
    double time_start = getTimeStamp();

    po::options_description desc("Smith-Waterman CUDA Usage");
    po::variables_map vm;
    
    try {
        desc.add_options()
            ("help", "Display this help message")
            ("query", po::value<std::string>()->required(),"Path to query file (required)")
            ("db", po::value<std::string>()->required(), "Path to database file (required)");

        po::store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);

        if(vm.count("help") || argc <= 1){
            cout << desc;
            return 1;
        }
    } catch (const po::required_option & e) {
        cout << desc;
        return 1;
    }

    std::string querypath = vm["query"].as<std::string>();
    FASTAQuery query(querypath, true);
    cout << "Input buffer:";
    query.print_buffer();
    cout << endl;
    string querySequenceBuffer = query.get_buffer();
    char querySequence[querySequenceBuffer.length() + 1];
	//querySequenceBuffer.copy(querySequence, querySequenceBuffer.length() + 1);
	copy(querySequenceBuffer.begin(), querySequenceBuffer.end(), querySequence);
	querySequence[querySequenceBuffer.length()] = '\0';

    // Parse database file
	std::string datapath = vm["db"].as<std::string>();
    FASTADatabase db(datapath);

	//cout << "Largest subject: " << largestSubjectLength << endl;
    //cout << "Num subjects: " << numSubjects << endl;
    //cout << "Accumulated db length: " << subjectLengthSum << endl;

    // alloc memory on GPU
    //float* d_input_query = new float[strlen(querySequence)];
	//memset(d_input_query, 0, sizeof(float) * strlen(querySequence));

    char* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (db.largestSubjectLength * db.numSubjects) * sizeof(char));
	/*
	memcpy(d_input_subject, subjectSequences[0], ((largestSubjectLength * numSubjects) + 1) * sizeof(char));
	
	for (int i = 0; i < numSubjects; i++) {
		strcat(d_input_subject, subjectSequences[i]);
	}
	*/

	// Set up offsets 
    int grid_y_dim = ceil(db.numSubjects / BLOCK_Y_DIM);
    
    char* d_input_offsets;
    hipMallocManaged((void**) &d_input_offsets, grid_y_dim * sizeof(char));

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((strlen(querySequence) + 1) * (db.largestSubjectLength + 1) * db.numSubjects) * sizeof(float));

	float* d_output_max_score;
    hipMallocManaged((void**) &d_output_max_score, db.numSubjects * sizeof(float));

	for (int i = 0; i < db.numSubjects; i++) {
        for (int j = 0; j < db.largestSubjectLength; j++) { // Will need to pad here
            if (j < db.subjectSequences[i].sequence.length()) {
                d_input_subject[i*db.largestSubjectLength + j] = convertStringToChar(db.subjectSequences[i].sequence[j]);
            }
            else d_input_subject[i*db.largestSubjectLength + j] = STAR;
        }
    }

	hipMemcpyToSymbol(HIP_SYMBOL(constQuery), querySequence, sizeof(char)*strlen(querySequence));
	hipMemcpyToSymbol(HIP_SYMBOL(constSubstitutionMatrix), blosum50_alpha, sizeof(int)*625);

    // Call GPU
    dim3 block(1, BLOCK_Y_DIM);
    dim3 grid(1, grid_y_dim);
 
    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, d_output_max_score, db.largestSubjectLength, strlen(querySequence), db.numSubjects);

    hipDeviceSynchronize();

	/*
    // Print results for 1 subject query
    for (int subject = 0; subject < 32; subject++) {
        char* seqA = querySequence;
        char* seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (strlen(seqB) + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (strlen(seqA) + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (strlen(seqB) + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (strlen(querySequence) + 1) * subject) + (i * (strlen(seqB) + 1)) + j] << " ";
            }
            cout << endl;
        }
    }*/

	vector<seqid_score> scores;
	for (int subject = 0; subject < db.numSubjects; subject++) {
        scores.push_back(make_pair(db.subjectSequences[subject].id, d_output_max_score[subject])); // change this
    }

	for (vector<seqid_score>::iterator it = scores.begin(); it != scores.end(); ++it) {
        cout << (*it).first << ":" << (*it).second << endl;
    }

    double time_end = getTimeStamp();
    double seconds_elapsed = time_end - time_start;

    cout << std::string(80, '=') << endl;
    cout << "METRICS:" << endl;
    cout << "Query length: " << strlen(querySequence) << " chars." << endl;
	cout << "Num subjects: " << db.numSubjects << endl;
    cout << "Sum of DB length: " << db.subjectLengthSum << " chars." << endl;
    cout << "Time elapsed: " << seconds_elapsed << " seconds." << endl;
    cout << "Performance: " << 1E-9 * (strlen(querySequence) * db.subjectLengthSum)
            / seconds_elapsed << " GCUPS." << endl;

    //delete[] d_input_query;

    // Free device memory
    //hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_output_scoring);
    hipFree(d_output_max_score);
    hipDeviceReset();
}
