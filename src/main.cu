
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <sstream>
#include <algorithm>

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define A 1
#define G 2
#define C 3
#define T 4

using namespace std;

__constant__ float constQuery[1024];

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequence(float* subject, float* scoringMatrix, int width, int height) {
    int substitutionMatrix[2] = {SEQ_EQUAL, SEQ_DIFF};

    register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

    float maxScore = 0;
    for (int i = 1; i < (height + 1); i++) {
        for (int j = 1; j < (width + 1); j++) {
            float score = 0;

            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);

            int similarityScore = 0;

            // Just index scoring matrix from shared/constant memory in the future
            if (constQuery[i - 1] == subject[width*yIndex + j - 1]) similarityScore = substitutionMatrix[0];
            else similarityScore = substitutionMatrix[1];

            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);

            maxScore = max(maxScore, score);

            scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
        }
    }
}

class ParsedFASTA {
private:
    bool isQuery;
    stringstream header;
    string buffer;
public:
    ParsedFASTA(const char *filepath, bool _isQuery) {
        isQuery = _isQuery;

        ifstream filestream;
        filestream.open(filepath);
        filestream.ignore(numeric_limits<streamsize>::max(), '\n');

        stringstream fasta_stream;
        fasta_stream << filestream.rdbuf();
        buffer.reserve(10000); // optimization -- reserve some arbitrary length
        string tmp;
        while (fasta_stream) {
            fasta_stream >> tmp;
            buffer.append(tmp);
        }
        filestream.close();
    };

    ~ParsedFASTA() {
    };

    void print_buffer() {
        cout << buffer << endl;
    };

    string get_buffer() {
        return buffer;
    };
};

int main( int argc, char *argv[] ) {
    double time_start = getTimeStamp();

    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }

    //string querySequence = argv[1];

    ParsedFASTA query(argv[1], true);
    cout << "Input buffer:";
    query.print_buffer();
    cout << endl;
    string querySequence = query.get_buffer();

    // Parse query file
    ifstream datafile;
    datafile.open(argv[2]);

    int subjectLengthSum = 0;

    string temp;
    vector<string> subjectSequences;
    while (datafile >> temp) {
       subjectSequences.push_back(temp);
       subjectLengthSum += temp.length();
    }

    // Just do the first 32 elements for a test
    int largestSubjectLength = subjectSequences[31].length();

    datafile.close();

    // alloc memory on GPU
    float* d_input_query = new float[querySequence.length()];
    memset(d_input_query, 0, sizeof(float) * querySequence.length());

    float* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (largestSubjectLength * 32) * sizeof(float));

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((querySequence.length() + 1) * (largestSubjectLength + 1) * 32) * sizeof(float));

    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < querySequence.length();i++) { // Pad to nearest 8 eventually here
        switch(querySequence[i])
        {
            case 'A': { d_input_query[i] = A;
                        break;
                    }
            case 'G': { d_input_query[i] = G;
                        break;
                    }
            case 'C': { d_input_query[i] = C;
                        break;
                    }
            case 'T': { d_input_query[i] = T;
                        break;
                    }
        }
    }

    for (int i = 0; i < 32; i++) {
        for (int j = 0; j < largestSubjectLength; j++) { // Will need to pad here
            switch(subjectSequences[i][j])
            {
                case 'A': { d_input_subject[i*largestSubjectLength + j] = A;
                            break;
                        }
                case 'G': { d_input_subject[i*largestSubjectLength + j] = G;
                            break;
                        }
                case 'C': { d_input_subject[i*largestSubjectLength + j] = C;
                            break;
                        }
                case 'T': { d_input_subject[i*largestSubjectLength + j] = T;
                            break;
                        }
            }
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(constQuery), d_input_query, sizeof(float)*querySequence.length());

    // Call GPU
    dim3 block(1, 32);
    dim3 grid(1, 1);

    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, largestSubjectLength, querySequence.length());

    hipDeviceSynchronize();

    // Print results for 1 subject query
    for (int subject = 0; subject < 32; subject++) {
        string seqA = querySequence;
        string seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (seqB.length() + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (seqA.length() + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (seqB.length() + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (querySequence.length() + 1) * subject) + (i * (seqB.length() + 1)) + j] << " ";
            }
            cout << endl;
        }
    }

    double time_end = getTimeStamp();
    double seconds_elapsed = time_end - time_start;

    std::cout << std::string(80, '=') << std::endl;
    cout << "METRICS:" << endl;
    cout << "Query length: " << querySequence.length() << " chars." << endl;
    cout << "Sum of DB length: " << subjectLengthSum << " chars." << endl;
    cout << "Time elapsed: " << seconds_elapsed << " seconds." << endl;
    cout << "Performance: " << 1E-9 * (querySequence.length() * subjectLengthSum)
            / seconds_elapsed << " GCUPS." << endl;

    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_output_scoring);
    hipDeviceReset();
}
