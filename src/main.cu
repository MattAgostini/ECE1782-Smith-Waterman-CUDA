#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <math.h>
#include <sstream>
#include <algorithm>
#include <map>
#include <set>
#include "boost/program_options.hpp"

#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define LENGTH_THRESHOLD 100

#define A 0
#define R 1
#define N 2
#define D 3
#define C 4
#define Q 5
#define E 6
#define G 7
#define H 8
#define I 9
#define L 10
#define K 11
#define M 12
#define F 13
#define P 14
#define S 15
#define T 16
#define W 17
#define Y 18
#define V 19
#define B 20
#define J 21
#define Z 22
#define X 23
#define STAR 24

#define BLOCK_Y_DIM 32.0

int blosum50[25][25] = {
//        A  R  N  D  C  Q  E  G  H  I  L  K  M  F  P  S  T  W  Y  V  B  J  Z  X  *
/* A */ { 5,-2,-1,-2,-1,-1,-1, 0,-2,-1,-2,-1,-1,-3,-1, 1, 0,-3,-2, 0,-2,-2,-1,-1,-5},
/* R */ {-2, 7,-1,-2,-4, 1, 0,-3, 0,-4,-3, 3,-2,-3,-3,-1,-1,-3,-1,-3,-1,-3, 0,-1,-5},
/* N */ {-1,-1, 7, 2,-2, 0, 0, 0, 1,-3,-4, 0,-2,-4,-2, 1, 0,-4,-2,-3, 5,-4, 0,-1,-5},
/* D */ {-2,-2, 2, 8,-4, 0, 2,-1,-1,-4,-4,-1,-4,-5,-1, 0,-1,-5,-3,-4, 6,-4, 1,-1,-5 },
/* C */ {-1,-4,-2,-4,13,-3,-3,-3,-3,-2,-2,-3,-2,-2,-4,-1,-1,-5,-3,-1,-3,-2,-3,-1,-5 }, 
/* Q */ {-1, 1, 0, 0,-3, 7, 2,-2, 1,-3,-2, 2, 0,-4,-1, 0,-1,-1,-1,-3, 0,-3, 4,-1,-5 },  
/* E */ {-1, 0, 0, 2,-3, 2, 6,-3, 0,-4,-3, 1,-2,-3,-1,-1,-1,-3,-2,-3, 1,-3, 5,-1,-5 }, 
/* G */ { 0,-3, 0,-1,-3,-2,-3, 8,-2,-4,-4,-2,-3,-4,-2, 0,-2,-3,-3,-4,-1,-4,-2,-1,-5 },
/* H */ {-2, 0, 1,-1,-3, 1, 0,-2,10,-4,-3, 0,-1,-1,-2,-1,-2,-3, 2,-4, 0,-3, 0,-1,-5 },
/* I */ {-1,-4,-3,-4,-2,-3,-4,-4,-4, 5, 2,-3, 2, 0,-3,-3,-1,-3,-1, 4,-4, 4,-3,-1,-5 },
/* L */ {-2,-3,-4,-4,-2,-2,-3,-4,-3, 2, 5,-3, 3, 1,-4,-3,-1,-2,-1, 1,-4, 4,-3,-1,-5 },
/* K */ {-1, 3, 0,-1,-3, 2, 1,-2, 0,-3,-3, 6,-2,-4,-1, 0,-1,-3,-2,-3, 0,-3, 1,-1,-5 },
/* M */ {-1,-2,-2,-4,-2, 0,-2,-3,-1, 2, 3,-2, 7, 0,-3,-2,-1,-1, 0, 1,-3, 2,-1,-1,-5 },
/* F */ {-3,-3,-4,-5,-2,-4,-3,-4,-1, 0, 1,-4, 0, 8,-4,-3,-2, 1, 4,-1,-4, 1,-4,-1,-5 },
/* P */ {-1,-3,-2,-1,-4,-1,-1,-2,-2,-3,-4,-1,-3,-4,10,-1,-1,-4,-3,-3,-2,-3,-1,-1,-5 },
/* S */ { 1,-1, 1, 0,-1, 0,-1, 0,-1,-3,-3, 0,-2,-3,-1, 5, 2,-4,-2,-2, 0,-3, 0,-1,-5 },
/* T */ { 0,-1, 0,-1,-1,-1,-1,-2,-2,-1,-1,-1,-1,-2,-1, 2, 5,-3,-2, 0, 0,-1,-1,-1,-5 },
/* W */ {-3,-3,-4,-5,-5,-1,-3,-3,-3,-3,-2,-3,-1, 1,-4,-4,-3,15, 2,-3,-5,-2,-2,-1,-5 },
/* Y */ {-2,-1,-2,-3,-3,-1,-2,-3, 2,-1,-1,-2, 0, 4,-3,-2,-2, 2, 8,-1,-3,-1,-2,-1,-5 },
/* V */ { 0,-3,-3,-4,-1,-3,-3,-4,-4, 4, 1,-3, 1,-1,-3,-2, 0,-3,-1, 5,-3, 2,-3,-1,-5 },
/* B */ {-2,-1, 5, 6,-3, 0, 1,-1, 0,-4,-4, 0,-3,-4,-2, 0, 0,-5,-3,-3, 6,-4, 1,-1,-5 },
/* J */ {-2,-3,-4,-4,-2,-3,-3,-4,-3, 4, 4,-3, 2, 1,-3,-3,-1,-2,-1, 2,-4, 4,-3,-1,-5 },
/* Z */ {-1, 0, 0, 1,-3, 4, 5,-2, 0,-3,-3, 1,-1,-4,-1, 0,-1,-2,-2,-3, 1,-3, 5,-1,-5 }, 
/* X */ {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-5 },
/* * */ {-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5 }
};

using namespace std;
namespace po = boost::program_options;

__constant__ float constQuery[1024];
__constant__ int constSubstitutionMatrix[625];

float convertStringToFloat(char character) {
    switch(character)
    {
        case 'A': { return A; }
        case 'R': { return R; }
        case 'N': { return N; }
        case 'D': { return D; }
        case 'C': { return C; }
        case 'Q': { return Q; }
        case 'E': { return E; }
        case 'G': { return G; }
        case 'H': { return H; }
        case 'I': { return I; }
        case 'L': { return L; }
        case 'K': { return K; }
        case 'M': { return M; }
        case 'F': { return F; }
        case 'P': { return P; }
        case 'S': { return S; }
        case 'T': { return T; }
        case 'W': { return W; }
        case 'Y': { return Y; }
        case 'V': { return V; }
        case 'B': { return B; }
        case 'J': { return J; }
        case 'Z': { return Z; }
        case 'X': { return X; }
    }
    return STAR;
}

struct subject_sequence {
    int id;
    string sequence;
};

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequence(float* subject, float* scoringMatrix, float* maxScoreList, 
                int width /*largestSubjectLength*/, int height /*querySequence.length()*/, int numSubjects) {
    

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (yIndex == 0) {
        //printf("GPU: %d %f\n", constSubstitutionMatrix[24], constQuery[0]);
    }
    
    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);

                int similarityScore = constSubstitutionMatrix[((int)constQuery[i - 1] * 25) + (int)subject[width*yIndex + j - 1]];
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);

                maxScore = max(maxScore, score);

                scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequenceCoalesced(float* subject, float* scoringMatrix, float* maxScoreList, 
                int width /*largestSubjectLength*/, int height /*querySequence.length()*/, int numSubjects) {
    
    int substitutionMatrix[2] = {3, -3};

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use map for different offsets (Change the width)
    int blockOffset = (blockIdx.y * blockDim.y)*(width + 1)*(height + 1);

    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * i)] - GAP_PENALTY);
                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] - GAP_PENALTY);

                int similarityScore = 0;

                if (constQuery[i - 1] == subject[threadIdx.y + ((j - 1) * blockDim.y)]) similarityScore = substitutionMatrix[0];
                else similarityScore = substitutionMatrix[1];

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] + similarityScore);

                maxScore = max(maxScore, score);
                
                scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * i)] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

class ParsedFASTA {
private:
    bool isQuery;
    stringstream header;
    string buffer;
public:
    ParsedFASTA(std::string filepath, bool _isQuery) {
        isQuery = _isQuery;

        ifstream filestream;
        filestream.open(filepath.c_str());

        buffer.reserve(10000); // optimization -- reserve some arbitrary length
        string tmp;
        getline(filestream, tmp); // Skip first line
        while (getline(filestream, tmp)) {
            buffer.append(tmp);
        }
        filestream.close();
    };

    ~ParsedFASTA() {
    };

    void print_buffer() {
        cout << buffer << endl;
    };

    string get_buffer() {
        return buffer;
    };
};

int main( int argc, char *argv[] ) {
    double time_start = getTimeStamp();

    po::options_description desc("Smith-Waterman CUDA Usage");
    po::variables_map vm;
    
    try {
        desc.add_options()
            ("help", "Display this help message")
            ("query", po::value<std::string>()->required(),"Path to query file (required)")
            ("db", po::value<std::string>()->required(), "Path to database file (required)");

        po::store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);

        if(vm.count("help") || argc <= 1){
            cout << desc;
            return 1;
        }
    } catch (const po::required_option & e) {
        cout << desc;
        return 1;
    }

    std::string querypath = vm["query"].as<std::string>();
    ParsedFASTA query(querypath, true);
    cout << "Input buffer:";
    query.print_buffer();
    cout << endl;
    string querySequence = query.get_buffer();

    // Parse database file
    ifstream databaseFile;
    std::string datapath = vm["db"].as<std::string>();
    databaseFile.open(datapath.c_str());

    int subjectLengthSum = 0;

    string temp;

    // key is sequence length, value is a vector of subject_sequence struct
    map<int, vector<subject_sequence> > parsedDB;

    vector<string> subjectSequences;
    string subjectSequence = "";
    int largestSubjectLength = 0;
    int numSubjects = 0;
    bool isFirst = true;

    subject_sequence tmp;

    int _id = 0;

    while (getline(databaseFile, temp)) {
        
        // This line denotes the start of a sequence
        if (temp[0] == '>') {
            if (!isFirst) {
                if (subjectSequence.length() <= LENGTH_THRESHOLD) {
                    tmp.id = _id++;
                    tmp.sequence = subjectSequence;
                    parsedDB[subjectSequence.length()].push_back(tmp);

                    subjectSequences.push_back(subjectSequence);
                    subjectLengthSum += subjectSequence.length();
                    largestSubjectLength = max(largestSubjectLength, (int)subjectSequence.length());
                    
                    numSubjects++;
                }
            }
            isFirst = false;
            
            //cout << subjectSequence << endl;
            subjectSequence = "";
        }
        else {
            subjectSequence += temp;
        }
        
    }
    // Adding last sequence 
    if (subjectSequence.length() <= LENGTH_THRESHOLD) {
        tmp.id = _id++;
        tmp.sequence = subjectSequence;
        parsedDB[subjectSequence.length()].push_back(tmp);

        subjectSequences.push_back(subjectSequence);
        subjectLengthSum += subjectSequence.length();
        largestSubjectLength = max(largestSubjectLength, (int)subjectSequence.length());
        
        numSubjects++;
    }


    databaseFile.close();

    /*
    for (map<int, vector<subject_sequence> >::iterator it = parsedDB.begin(); it != parsedDB.end(); ++it) {
        cout << it->first 
             << ":"
             << it->second.size()
             << endl;
    } */

    // alloc memory on GPU
    float* d_input_query = new float[querySequence.length()];
    memset(d_input_query, 0, sizeof(float) * querySequence.length());

    float* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (largestSubjectLength * numSubjects) * sizeof(float));
    
    // Set up offsets 
    int grid_y_dim = ceil(numSubjects / BLOCK_Y_DIM);
    
    float* d_input_offsets;
    hipMallocManaged((void**) &d_input_offsets, grid_y_dim * sizeof(float));

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((querySequence.length() + 1) * (largestSubjectLength + 1) * numSubjects) * sizeof(float));
    
    float* d_output_max_score;
    hipMallocManaged((void**) &d_output_max_score, numSubjects * sizeof(float));

    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < querySequence.length();i++) { // Pad to nearest 8 eventually here
        d_input_query[i] = convertStringToFloat(querySequence[i]);
    }
    /*
    int blockPop = 0;
    int blockNum = 1;
    int blockWidth = 0;
    for (map<int, vector<subject_sequence> >::reverse_iterator it = parsedDB.rbegin(); it != parsedDB.rend(); ++it) {
        blockWidth = max(blockWidth, it->first);
        for (int i = 0; i < it->second.size(); ++i) {
            if (blockPop >= BLOCK_Y_DIM) {
                blockPop = 0;
                d_input_offsets[blockNum] = d_input_offsets[blockNum - 1] + (BLOCK_Y_DIM * blockWidth); // Need to include the query length for scoring matrix
                blockNum++;
                blockWidth = it->first;
            }
            
            
            blockPop++;
        }
    }
    */
    for (int i = 0; i < numSubjects; i++) {
        for (int j = 0; j < largestSubjectLength; j++) { // Will need to pad here
            if (j < subjectSequences[i].length()) {
                d_input_subject[i*largestSubjectLength + j] = convertStringToFloat(subjectSequences[i][j]);
            }
            else d_input_subject[i*largestSubjectLength + j] = STAR;
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(constQuery), d_input_query, sizeof(float)*querySequence.length());
    hipMemcpyToSymbol(HIP_SYMBOL(constSubstitutionMatrix), blosum50, sizeof(int)*625);
    
    // Call GPU
    dim3 block(1, BLOCK_Y_DIM);
    dim3 grid(1, grid_y_dim);
    
    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, d_output_max_score, largestSubjectLength, querySequence.length(), numSubjects);

    hipDeviceSynchronize();

    /*
    // Print results for 1 subject query
    for (int subject = 0; subject < numSubjects; subject++) {
        string seqA = querySequence;
        string seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (seqB.length() + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (seqA.length() + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (seqB.length() + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (querySequence.length() + 1) * subject) + (i * (seqB.length() + 1)) + j] << " ";
            }
            cout << endl;
        }
    }
    */
    
    // Print results for 1 subject query
    for (int subject = 0; subject < numSubjects; subject++) {
        cout << d_output_max_score[subject] << endl;
    }

    double time_end = getTimeStamp();
    double seconds_elapsed = time_end - time_start;

    cout << std::string(80, '=') << endl;
    cout << "METRICS:" << endl;
    cout << "Query length: " << querySequence.length() << " chars." << endl;
    cout << "Num subjects: " << numSubjects << endl;
    cout << "Sum of DB length: " << subjectLengthSum << " chars." << endl;
    cout << "Time elapsed: " << seconds_elapsed << " seconds." << endl;
    cout << "Performance: " << 1E-9 * (querySequence.length() * subjectLengthSum)
            / seconds_elapsed << " GCUPS." << endl;

    delete[] d_input_query;

    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_input_offsets);
    hipFree(d_output_scoring);
    hipFree(d_output_max_score);
    hipDeviceReset();
}
