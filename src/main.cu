#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <limits>
#include <math.h>
#include <sstream>
#include <algorithm>
#include "boost/program_options.hpp"

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define A 1
#define G 2
#define C 3
#define T 4

using namespace std;
namespace po = boost::program_options;

__constant__ float constQuery[1024];

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequence(float* subject, float* scoringMatrix, float* maxScoreList, int width, int height, int numSubjects) {
    int substitutionMatrix[2] = {SEQ_EQUAL, SEQ_DIFF};

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);

                int similarityScore = 0;

                // Just index scoring matrix from shared/constant memory in the future
                if (constQuery[i - 1] == subject[width*yIndex + j - 1]) similarityScore = substitutionMatrix[0];
                else similarityScore = substitutionMatrix[1];

                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);

                maxScore = max(maxScore, score);

                scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

class ParsedFASTA {
private:
    bool isQuery;
    stringstream header;
    string buffer;
public:
    ParsedFASTA(std::string filepath, bool _isQuery) {
        isQuery = _isQuery;

        ifstream filestream;
        filestream.open(filepath.c_str());
        filestream.ignore(numeric_limits<streamsize>::max(), '\n');

        stringstream fasta_stream;
        fasta_stream << filestream.rdbuf();
        buffer.reserve(10000); // optimization -- reserve some arbitrary length
        string tmp;
        while (fasta_stream) {
            fasta_stream >> tmp;
            buffer.append(tmp);
        }
        filestream.close();
    };

    ~ParsedFASTA() {
    };

    void print_buffer() {
        cout << buffer << endl;
    };

    string get_buffer() {
        return buffer;
    };
};

int main( int argc, char *argv[] ) {
    double time_start = getTimeStamp();

    po::options_description desc("Smith-Waterman CUDA Usage");
    po::variables_map vm;
    
    try {
        desc.add_options()
            ("help", "Display this help message")
            ("query", po::value<std::string>()->required(),"Path to query file (required)")
            ("db", po::value<std::string>()->required(), "Path to database file (required)");

        po::store(po::parse_command_line(argc, argv, desc), vm);
        po::notify(vm);

        if(vm.count("help") || argc <= 1){
            cout << desc;
            return 1;
        }
    } catch (const po::required_option & e) {
        cout << desc;
        return 1;
    }

    std::string querypath = vm["query"].as<std::string>();
    ParsedFASTA query(querypath, true);
    cout << "Input buffer:";
    query.print_buffer();
    cout << endl;
    string querySequence = query.get_buffer();

    // Parse database file
    ifstream databaseFile;
    std::string datapath = vm["db"].as<std::string>();
    databaseFile.open(datapath.c_str());

    int subjectLengthSum = 0;

    string temp;
    vector<string> subjectSequences;
    int count = 0;
    int largestSubjectLength = 0;
    int numSubjects = 0;
    while (databaseFile >> temp && count < 32) {
        
        if (temp.find("SQ") != string::npos) {
            
            databaseFile >> temp; // Skip "SEQUENCE"
            
            int length = 0;
            databaseFile >> length; // Extract sequence length
            largestSubjectLength = max(largestSubjectLength, length);
            
            // Skipping rest of line
            databaseFile >> temp;
            databaseFile >> temp;
            databaseFile >> temp;
            databaseFile >> temp;
            databaseFile >> temp;
            
            // Start processing sequence
            string subjectSequence = "";
            for (int i = 0; i < length; i += 10) {
                databaseFile >> temp;
                subjectSequence += temp;
            }
            
            //cout << subjectSequence << endl;
            
            //count++;
            numSubjects++;
            
            subjectSequences.push_back(subjectSequence);
            subjectLengthSum += subjectSequence.length();
        }
    }

    databaseFile.close();

    // alloc memory on GPU
    float* d_input_query = new float[querySequence.length()];
    memset(d_input_query, 0, sizeof(float) * querySequence.length());

    float* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (largestSubjectLength * numSubjects) * sizeof(float));

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((querySequence.length() + 1) * (largestSubjectLength + 1) * numSubjects) * sizeof(float));
    
    float* d_output_max_score;
    hipMallocManaged((void**) &d_output_max_score, numSubjects * sizeof(float));

    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < querySequence.length();i++) { // Pad to nearest 8 eventually here
        switch(querySequence[i])
        {
            case 'A': { d_input_query[i] = A;
                        break;
                    }
            case 'G': { d_input_query[i] = G;
                        break;
                    }
            case 'C': { d_input_query[i] = C;
                        break;
                    }
            case 'T': { d_input_query[i] = T;
                        break;
                    }
        }
    }

    for (int i = 0; i < numSubjects; i++) {
        for (int j = 0; j < largestSubjectLength; j++) { // Will need to pad here
            switch(subjectSequences[i][j])
            {
                case 'A': { d_input_subject[i*largestSubjectLength + j] = A;
                            break;
                        }
                case 'G': { d_input_subject[i*largestSubjectLength + j] = G;
                            break;
                        }
                case 'C': { d_input_subject[i*largestSubjectLength + j] = C;
                            break;
                        }
                case 'T': { d_input_subject[i*largestSubjectLength + j] = T;
                            break;
                        }
            }
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(constQuery), d_input_query, sizeof(float)*querySequence.length());

    int grid_y_dim = ceil(numSubjects / 32.0);
    
    // Call GPU
    dim3 block(1, 32);
    dim3 grid(1, grid_y_dim);
    
    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, d_output_max_score, largestSubjectLength, querySequence.length(), numSubjects);

    hipDeviceSynchronize();

    /*
    // Print results for 1 subject query
    for (int subject = 0; subject < numSubjects; subject++) {
        string seqA = querySequence;
        string seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (seqB.length() + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (seqA.length() + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (seqB.length() + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (querySequence.length() + 1) * subject) + (i * (seqB.length() + 1)) + j] << " ";
            }
            cout << endl;
        }
    }
    */
    
    // Print results for 1 subject query
    for (int subject = 0; subject < numSubjects; subject++) {
        cout << d_output_max_score[subject] << endl;
    }
    

    double time_end = getTimeStamp();
    double seconds_elapsed = time_end - time_start;

    cout << std::string(80, '=') << endl;
    cout << "METRICS:" << endl;
    cout << "Query length: " << querySequence.length() << " chars." << endl;
    cout << "Sum of DB length: " << subjectLengthSum << " chars." << endl;
    cout << "Time elapsed: " << seconds_elapsed << " seconds." << endl;
    cout << "Performance: " << 1E-9 * (querySequence.length() * subjectLengthSum)
            / seconds_elapsed << " GCUPS." << endl;

    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_output_scoring);
    hipFree(d_output_max_score);
    hipDeviceReset();
}
