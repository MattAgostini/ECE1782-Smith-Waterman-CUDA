#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>

#include "FASTAParsers.h"
#include "SWSolver.h"

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define LENGTH_THRESHOLD 100

#define A 0
#define R 1
#define N 2
#define D 3
#define C 4
#define Q 5
#define E 6
#define G 7
#define H 8
#define I 9
#define L 10
#define K 11
#define M 12
#define F 13
#define P 14
#define S 15
#define T 16
#define W 17
#define Y 18
#define V 19
#define B 20
#define J 21
#define Z 22
#define X 23
#define STAR 24

#define BLOCK_Y_DIM 32.0

// first is sequence ID, second is max score


int blosum50[25][25] = {
//        A  R  N  D  C  Q  E  G  H  I  L  K  M  F  P  S  T  W  Y  V  B  J  Z  X  *
/* A */ { 5,-2,-1,-2,-1,-1,-1, 0,-2,-1,-2,-1,-1,-3,-1, 1, 0,-3,-2, 0,-2,-2,-1,-1,-5},
/* R */ {-2, 7,-1,-2,-4, 1, 0,-3, 0,-4,-3, 3,-2,-3,-3,-1,-1,-3,-1,-3,-1,-3, 0,-1,-5},
/* N */ {-1,-1, 7, 2,-2, 0, 0, 0, 1,-3,-4, 0,-2,-4,-2, 1, 0,-4,-2,-3, 5,-4, 0,-1,-5},
/* D */ {-2,-2, 2, 8,-4, 0, 2,-1,-1,-4,-4,-1,-4,-5,-1, 0,-1,-5,-3,-4, 6,-4, 1,-1,-5 },
/* C */ {-1,-4,-2,-4,13,-3,-3,-3,-3,-2,-2,-3,-2,-2,-4,-1,-1,-5,-3,-1,-3,-2,-3,-1,-5 }, 
/* Q */ {-1, 1, 0, 0,-3, 7, 2,-2, 1,-3,-2, 2, 0,-4,-1, 0,-1,-1,-1,-3, 0,-3, 4,-1,-5 },  
/* E */ {-1, 0, 0, 2,-3, 2, 6,-3, 0,-4,-3, 1,-2,-3,-1,-1,-1,-3,-2,-3, 1,-3, 5,-1,-5 }, 
/* G */ { 0,-3, 0,-1,-3,-2,-3, 8,-2,-4,-4,-2,-3,-4,-2, 0,-2,-3,-3,-4,-1,-4,-2,-1,-5 },
/* H */ {-2, 0, 1,-1,-3, 1, 0,-2,10,-4,-3, 0,-1,-1,-2,-1,-2,-3, 2,-4, 0,-3, 0,-1,-5 },
/* I */ {-1,-4,-3,-4,-2,-3,-4,-4,-4, 5, 2,-3, 2, 0,-3,-3,-1,-3,-1, 4,-4, 4,-3,-1,-5 },
/* L */ {-2,-3,-4,-4,-2,-2,-3,-4,-3, 2, 5,-3, 3, 1,-4,-3,-1,-2,-1, 1,-4, 4,-3,-1,-5 },
/* K */ {-1, 3, 0,-1,-3, 2, 1,-2, 0,-3,-3, 6,-2,-4,-1, 0,-1,-3,-2,-3, 0,-3, 1,-1,-5 },
/* M */ {-1,-2,-2,-4,-2, 0,-2,-3,-1, 2, 3,-2, 7, 0,-3,-2,-1,-1, 0, 1,-3, 2,-1,-1,-5 },
/* F */ {-3,-3,-4,-5,-2,-4,-3,-4,-1, 0, 1,-4, 0, 8,-4,-3,-2, 1, 4,-1,-4, 1,-4,-1,-5 },
/* P */ {-1,-3,-2,-1,-4,-1,-1,-2,-2,-3,-4,-1,-3,-4,10,-1,-1,-4,-3,-3,-2,-3,-1,-1,-5 },
/* S */ { 1,-1, 1, 0,-1, 0,-1, 0,-1,-3,-3, 0,-2,-3,-1, 5, 2,-4,-2,-2, 0,-3, 0,-1,-5 },
/* T */ { 0,-1, 0,-1,-1,-1,-1,-2,-2,-1,-1,-1,-1,-2,-1, 2, 5,-3,-2, 0, 0,-1,-1,-1,-5 },
/* W */ {-3,-3,-4,-5,-5,-1,-3,-3,-3,-3,-2,-3,-1, 1,-4,-4,-3,15, 2,-3,-5,-2,-2,-1,-5 },
/* Y */ {-2,-1,-2,-3,-3,-1,-2,-3, 2,-1,-1,-2, 0, 4,-3,-2,-2, 2, 8,-1,-3,-1,-2,-1,-5 },
/* V */ { 0,-3,-3,-4,-1,-3,-3,-4,-4, 4, 1,-3, 1,-1,-3,-2, 0,-3,-1, 5,-3, 2,-3,-1,-5 },
/* B */ {-2,-1, 5, 6,-3, 0, 1,-1, 0,-4,-4, 0,-3,-4,-2, 0, 0,-5,-3,-3, 6,-4, 1,-1,-5 },
/* J */ {-2,-3,-4,-4,-2,-3,-3,-4,-3, 4, 4,-3, 2, 1,-3,-3,-1,-2,-1, 2,-4, 4,-3,-1,-5 },
/* Z */ {-1, 0, 0, 1,-3, 4, 5,-2, 0,-3,-3, 1,-1,-4,-1, 0,-1,-2,-2,-3, 1,-3, 5,-1,-5 }, 
/* X */ {-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-1,-5 },
/* * */ {-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5,-5 }
};

using namespace std;

__constant__ float constQuery[1024];
__constant__ int constSubstitutionMatrix[625];

float convertStringToFloat(char character) {
    switch(character)
    {
        case 'A': { return A; }
        case 'R': { return R; }
        case 'N': { return N; }
        case 'D': { return D; }
        case 'C': { return C; }
        case 'Q': { return Q; }
        case 'E': { return E; }
        case 'G': { return G; }
        case 'H': { return H; }
        case 'I': { return I; }
        case 'L': { return L; }
        case 'K': { return K; }
        case 'M': { return M; }
        case 'F': { return F; }
        case 'P': { return P; }
        case 'S': { return S; }
        case 'T': { return T; }
        case 'W': { return W; }
        case 'Y': { return Y; }
        case 'V': { return V; }
        case 'B': { return B; }
        case 'J': { return J; }
        case 'Z': { return Z; }
        case 'X': { return X; }
    }
    return STAR;
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequence(float* subject, float* scoringMatrix, float* maxScoreList, 
                int width /*largestSubjectLength*/, int height /*querySequence.length()*/, int numSubjects) {
    

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (yIndex == 0) {
        //printf("GPU: %d %f\n", constSubstitutionMatrix[24], constQuery[0]);
    }
    
    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);

                int similarityScore = constSubstitutionMatrix[((int)constQuery[i - 1] * 25) + (int)subject[width*yIndex + j - 1]];
                score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);

                maxScore = max(maxScore, score);

                scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

// Kernel function for computing the scoring matrix of a sequence
__global__ void f_scoreSequenceCoalesced(float* subject, float* scoringMatrix, float* maxScoreList, 
                int width /*largestSubjectLength*/, int height /*querySequence.length()*/, int numSubjects) {
    
    int substitutionMatrix[2] = {3, -3};

    //register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use map for different offsets (Change the width)
    int blockOffset = (blockIdx.y * blockDim.y)*(width + 1)*(height + 1);

    float maxScore = 0;
        if (yIndex < numSubjects) {
        for (int i = 1; i < (height + 1); i++) {
            for (int j = 1; j < (width + 1); j++) {
                float score = 0;

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * i)] - GAP_PENALTY);
                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] - GAP_PENALTY);

                int similarityScore = 0;

                if (constQuery[i - 1] == subject[threadIdx.y + ((j - 1) * blockDim.y)]) similarityScore = substitutionMatrix[0];
                else similarityScore = substitutionMatrix[1];

                score = max(score, scoringMatrix[blockOffset + (threadIdx.y + ((j - 1) * blockDim.y * (height + 1))) + (blockDim.y * (i - 1))] + similarityScore);

                maxScore = max(maxScore, score);
                
                scoringMatrix[blockOffset + (threadIdx.y + (j * blockDim.y * (height + 1))) + (blockDim.y * i)] = score;
            }
        }
        maxScoreList[yIndex] = maxScore;
    }
}

vector<seqid_score> smith_waterman_cuda(FASTAQuery &query, FASTADatabase &db) {
	string querySequence = query.get_buffer();
	vector<seqid_score> scores;
	
    // alloc memory on GPU
    float* d_input_query = new float[querySequence.length()];
    memset(d_input_query, 0, sizeof(float) * querySequence.length());

    float* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (db.largestSubjectLength * db.numSubjects) * sizeof(float));
    
    // Set up offsets 
    int grid_y_dim = ceil(db.numSubjects / BLOCK_Y_DIM);
    
    float* d_input_offsets;
    hipMallocManaged((void**) &d_input_offsets, grid_y_dim * sizeof(float));

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((querySequence.length() + 1) *
                (db.largestSubjectLength + 1) * db.numSubjects) * sizeof(float));
    
    float* d_output_max_score;
    hipMallocManaged((void**) &d_output_max_score, db.numSubjects * sizeof(float));

    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < querySequence.length();i++) { // Pad to nearest 8 eventually here
        d_input_query[i] = convertStringToFloat(querySequence[i]);
    }
	
    /*
    int blockPop = 0;
    int blockNum = 1;
    int blockWidth = 0;
    for (map<int, vector<subject_sequence> >::reverse_iterator it = parsedDB.rbegin(); it != parsedDB.rend(); ++it) {
        blockWidth = max(blockWidth, it->first);
        for (int i = 0; i < it->second.size(); ++i) {
            if (blockPop >= BLOCK_Y_DIM) {
                blockPop = 0;
                d_input_offsets[blockNum] = d_input_offsets[blockNum - 1] + (BLOCK_Y_DIM * blockWidth); // Need to include the query length for scoring matrix
                blockNum++;
                blockWidth = it->first;
            }
            
            
            blockPop++;
        }
    }
    */
	
    for (int i = 0; i < db.numSubjects; i++) {
        for (int j = 0; j < db.largestSubjectLength; j++) { // Will need to pad here
            if (j < db.subjectSequences[i].sequence.length()) {
                d_input_subject[i*db.largestSubjectLength + j] = convertStringToFloat(db.subjectSequences[i].sequence[j]);
            }
            else d_input_subject[i*db.largestSubjectLength + j] = STAR;
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(constQuery), d_input_query, sizeof(float)*querySequence.length());
    hipMemcpyToSymbol(HIP_SYMBOL(constSubstitutionMatrix), blosum50, sizeof(int)*625);
    
    // Call GPU
    dim3 block(1, BLOCK_Y_DIM);
    dim3 grid(1, grid_y_dim);
    
    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, d_output_max_score, db.largestSubjectLength, querySequence.length(), db.numSubjects);

    hipDeviceSynchronize();

    /*
    // Print results for 1 subject query
    for (int subject = 0; subject < numSubjects; subject++) {
        string seqA = querySequence;
        string seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (seqB.length() + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (seqA.length() + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (seqB.length() + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (querySequence.length() + 1) * subject) + (i * (seqB.length() + 1)) + j] << " ";
            }
            cout << endl;
        }
    }
    */
    
    // Print results for 1 subject query
	
	
    for (int subject = 0; subject < db.numSubjects; subject++) {
        scores.push_back(make_pair(db.subjectSequences[subject].id, d_output_max_score[subject])); // change this
    }
	
    delete[] d_input_query;

    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_input_offsets);
    hipFree(d_output_scoring);
    hipFree(d_output_max_score);
    hipDeviceReset();
	
    return scores;
}

