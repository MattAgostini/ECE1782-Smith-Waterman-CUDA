
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>
#include <iostream>
#include <numeric>
#include <fstream>
#include <vector>

#define SEQ_EQUAL 3
#define SEQ_DIFF -3
#define GAP_PENALTY 2
// define affine penalty ?

#define FROM_LEFT 1
#define FROM_TOP 2
#define FROM_TOP_LEFT 3

#define MAX_BLOCK_SIZE 1024
#define MAX_GRID_DIM 65535

#define A 1
#define G 2
#define C 3
#define T 4

using namespace std;

__constant__ char constQuery[1024];

// Time stamp function
double getTimeStamp() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

__global__ void f_scoreSequence(char* subject, float* scoringMatrix, int width, int height) {
    // Do the scoring
    int substitutionMatrix[2] = {SEQ_EQUAL, SEQ_DIFF};

    register int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    register int yIndex = threadIdx.y + blockIdx.y * blockDim.y;

    float maxScore = 0;
    for (int i = 1; i < (height + 1); i++) {
        for (int j = 1; j < (width + 1); j++) {
            float score = 0;

            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j - 1] - GAP_PENALTY);
            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j] - GAP_PENALTY);

            int similarityScore = 0;

            // Just index scoring matrix from shared/constant memory in the future
            if (constQuery[i - 1] == subject[width*yIndex + j - 1]) similarityScore = substitutionMatrix[0];
            else similarityScore = substitutionMatrix[1];

            score = max(score, scoringMatrix[(width + 1)*(height + 1)*yIndex + ((i - 1) * (width + 1)) + j - 1] + similarityScore);
            
            maxScore = max(maxScore, score);

            scoringMatrix[(width + 1)*(height + 1)*yIndex + (i * (width + 1)) + j] = score;
        }
    }
}

int main( int argc, char *argv[] ) {
    double time_start = getTimeStamp();

    // get program arguments
    if (argc != 3) {
        printf("Error: wrong number of args\n");
        exit(1);
    }

    char* querySequence = argv[1];

    // Parse query file
    ifstream datafile;
    datafile.open(argv[2]);

    int subjectLengthSum = 0;



    char* temp;
    vector<char*> subjectSequences;
    while (datafile >> temp) {
       subjectSequences.push_back(temp);
       subjectLengthSum += strlen(temp);
    }

    // Just do the first 32 elements for a test
    int largestSubjectLength = strlen(subjectSequences[31]);

    datafile.close();

    // alloc memory on GPU
    float* d_input_query = new float[strlen(querySequence)];
	memset(d_input_query, 0, sizeof(float) * strlen(querySequence));
 
    char* d_input_subject;
    hipMallocManaged((void**) &d_input_subject, (largestSubjectLength * 32) * sizeof(char));
	memcpy(d_input_subject, subjectSequences[0], ((largestSubjectLength * 32) + 1) * sizeof(char));
	
	for (int i = 1; i < subjectSequences.size(); i++)
		strcat(d_input_subject, subjectSequences[i]);

    float* d_output_scoring;
    hipMallocManaged((void**) &d_output_scoring, ((strlen(querySequence) + 1) * (largestSubjectLength + 1) * 32) * sizeof(float));

    // Convert string to float representation (can't really use strings on the GPU)
    for (int i = 0; i < strlen(querySequence);i++) {
        switch(querySequence[i])
        {
            case 'A': { d_input_query[i] = A;
                        break;
                    }
            case 'G': { d_input_query[i] = G;
                        break;
                    }
            case 'C': { d_input_query[i] = C;
                        break;
                    }
            case 'T': { d_input_query[i] = T;
                        break;
                    }
        }
    }
	/*
    for (int j = 0; j < 32; j++) {
        for (int i = 0; i < largestSubjectLength; i++) {
            switch(subjectSequences[j][i])
            {
                case 'A': { d_input_subject[i] = 'A';
                            break;
                        }
                case 'G': { d_input_subject[i] = 'G';
                            break;
                        }
                case 'C': { d_input_subject[i] = 'C';
                            break;
                        }
                case 'T': { d_input_subject[i] = 'T';
                            break;
                        }
            }
        }
    }
	*/
	hipMemcpyToSymbol(HIP_SYMBOL(constQuery), querySequence, sizeof(char)*strlen(querySequence));

    // Call GPU
    dim3 block(1, 32);
    dim3 grid(1, 1);
 
    f_scoreSequence<<<grid, block>>>(d_input_subject, d_output_scoring, largestSubjectLength, strlen(querySequence));

    hipDeviceSynchronize();

    // Print results for 1 subject query
    for (int subject = 0; subject < 32; subject++) {
        char* seqA = querySequence;
        char* seqB = subjectSequences[subject];

        cout << "    ";
        for (int j = 0; j < (strlen(seqB) + 1); j++) {
            cout << seqB[j] << " ";
        }
        cout << endl;

        for (int i = 0; i < (strlen(seqA) + 1); i++) {
            if (i != 0) cout << seqA[i - 1] << " ";
            else cout << "  ";
            for (int j = 0; j < (strlen(seqB) + 1); j++) {
                cout << d_output_scoring[((largestSubjectLength + 1) * (strlen(querySequence) + 1) * subject) + (i * (strlen(seqB) + 1)) + j] << " ";
            }
            cout << endl;
        }
    }

    double time_end = getTimeStamp();
    double seconds_elapsed = time_end - time_start;

    std::cout << std::string(80, '=') << std::endl;
    cout << "METRICS:" << endl;
    cout << "Query length: " << strlen(querySequence) << " chars." << endl;
    cout << "Sum of DB length: " << subjectLengthSum << " chars." << endl;
    cout << "Time elapsed: " << seconds_elapsed << " seconds." << endl;
    cout << "Performance: " << 1E-9 * (strlen(querySequence) * subjectLengthSum)
            / seconds_elapsed << " GCUPS." << endl;

    // Free device memory
    hipFree(d_input_query);
    hipFree(d_input_subject);
    hipFree(d_output_scoring);
    hipDeviceReset();
}
